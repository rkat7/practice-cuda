#include <stdio.h>

int main(){
	float *A_h, *A_d;
	hipMalloc((**void) &A_d, sizeof(float));
	cudaMemCpy(A_d, A_h, sizeof(float), cudaMemCpyHostToDevice);
	hipFree(A_d);
	return 0;
}


#just basic code to copy a floating point value from host to device and free it for giggles.TT